#include <cmath>
#include <iostream>
#include <fstream>
#include <sputnik/spmm/cuda_spmm.h>
#include <random>
#include <hip/hip_runtime_api.h> 
#include "hipblas.h"
#include "utils.h"

std::mt19937 gen;
std::random_device rd;

int main(int argc, char* argv[]) {
    gen = std::mt19937{rd()};

    std::string matrixPath;
    int sparsity;

    int M, N, K;

    float* valuesA;
    int* rowOffsetsA;
    int* colIndicesA;
    int nnz;

    if (argc == 2) {
        matrixPath = argv[1];
        readDLMCMatrix(matrixPath, valuesA, rowOffsetsA, colIndicesA, nnz, M, K);
    } else if (argc == 5) {
        M = std::stoi(argv[1]);
        K = std::stoi(argv[2]);
        N = std::stoi(argv[3]);;
        sparsity = std::stof(argv[4]);
        float* A = generateMatrix(M, K, sparsity);
        convertToCSR(valuesA, rowOffsetsA, colIndicesA, nnz, M, K, A);
        delete A;
    } else {
        std::cerr << "Invalid number of arguments." << std::endl;
    }

    N = M;
    B = generateMatrix(K, N, 0);

    float* d_A;
    float* d_B;
    float* d_C;

    float alpha = 1;
    float beta = 0;

    gpuErrchk(hipMalloc(&d_A, M * K * sizeof(float)));
    gpuErrchk(hipMalloc(&d_B, K * N * sizeof(float)));
    gpuErrchk(hipMalloc(&d_C, M * N * sizeof(float)));

    hipEvent_t start, stop;
    gpuErrchk(hipEventCreate(&start));
    gpuErrchk(hipEventCreate(&stop));

    gpuErrchk(hipEventRecord(start));
    status = hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, N, K, &alpha,
            d_A, HIP_R_32F, M, 
            d_B, HIP_R_32F, K, 
            &beta,
            d_C, HIP_R_32F, M, 
            HIPBLAS_COMPUTE_32F_FAST_TF32,
            CUBLAS_GEMM_DEFAULT_TENSOR_OP);

    if (status != HIPBLAS_STATUS_SUCCESS) {
        printf ("CUBLAS initialization failed\n");
        return EXIT_FAILURE;
    }

    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk(hipEventRecord(stop));
    gpuErrchk(hipEventSynchronize(stop));
    float milliseconds = 0;
    gpuErrchk(hipEventElapsedTime(&milliseconds, start, stop));

    std::cout << milliseconds << std::endl;

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipblasDestroy(handle);

    delete A;
    delete B;
    delete valuesA;
    delete rowOffsetsA;
    delete colIndicesA;

    return 0;
}
