#include <cmath>
#include <string>
#include <iomanip>
#include <iostream>
#include <fstream>
#include <sputnik/spmm/cuda_spmm.h>
#include <random>
#include <hip/hip_runtime_api.h> 
#include <hipsparse.h>        
#include "utils.h"

std::mt19937 gen;
std::random_device rd;

int main(int argc, char* argv[]) {
    gen = std::mt19937{rd()};

    std::string matrixPath;
    int sparsity;

    int M, N, K;

    float* valuesA;
    int* rowOffsetsA;
    int* colIndicesA;
    int nnz;

    if (argc == 2) {
        matrixPath = argv[1];
        readDLMCMatrix(matrixPath, valuesA, rowOffsetsA, colIndicesA, nnz, M, K);
    } else if (argc == 5) {
        M = std::stoi(argv[1]);
        K = std::stoi(argv[2]);
        N = std::stoi(argv[3]);;
        sparsity = std::stof(argv[4]);
        float* A = generateMatrix(M, K, sparsity);
        convertToCSR(valuesA, rowOffsetsA, colIndicesA, nnz, M, K, A);
        delete A;
    } else {
        std::cerr << "Invalid number of arguments." << std::endl;
    }

    hipStream_t stream;
    hipStreamCreate(&stream);

    int* rowIndices = new int[M];

    for (int i = 0; i < M; ++i)
        rowIndices[i] = i;

    
    N = M;
    float* B = generateMatrix(K, N, 0);

    float* d_valuesA;
    gpuErrchk(hipMalloc(&d_valuesA, nnz * sizeof(float)));
    int* d_colIndicesA;
    gpuErrchk(hipMalloc(&d_colIndicesA, nnz * sizeof(int)));
    int* d_rowOffsetsA;
    gpuErrchk(hipMalloc(&d_rowOffsetsA, (M + 1) * sizeof(int)));

    float* d_B;
    gpuErrchk(hipMalloc(&d_B, K * N * sizeof(float)));
    
    int* d_rowIndices;
    gpuErrchk(hipMalloc(&d_rowIndices, M * sizeof(int)));

    gpuErrchk(hipMemcpy(d_valuesA, valuesA, nnz * sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_colIndicesA, colIndicesA, nnz * sizeof(int), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_rowOffsetsA, rowOffsetsA, (M + 1) * sizeof(int), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_B, B, K * N * sizeof(float), hipMemcpyHostToDevice));

    gpuErrchk(hipMemcpy(d_rowIndices, rowIndices, M * sizeof(int), hipMemcpyHostToDevice));

    float* d_C;
    gpuErrchk(hipMalloc(&d_C, M * N * sizeof(float)));

    hipEvent_t start, stop;
    gpuErrchk(hipEventCreate(&start));
    gpuErrchk(hipEventCreate(&stop));

    gpuErrchk(hipEventRecord(start));
    gpuErrchk(sputnik::CudaSpmm(M, K, N, nnz, d_rowIndices, d_valuesA, d_rowOffsetsA, d_colIndicesA, d_B, d_C, stream));
    gpuErrchk(hipEventRecord(stop));
    gpuErrchk(hipEventSynchronize(stop));
    float milliseconds = 0;
    gpuErrchk(hipEventElapsedTime(&milliseconds, start, stop));

    std::cout << milliseconds << std::endl;

    hipFree(d_valuesA);
    hipFree(d_colIndicesA);
    hipFree(d_rowOffsetsA);
    hipFree(d_B);
    hipFree(d_rowIndices);
    hipFree(d_C);

    delete B;
    delete valuesA;
    delete rowOffsetsA;
    delete colIndicesA;

    return 0;
}
