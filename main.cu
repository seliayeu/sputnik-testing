#include <cmath>
#include <iostream>
#include <fstream>
#include <sputnik/spmm/cuda_spmm.h>
#include <random>
#include <hip/hip_runtime_api.h> 
#include <hipsparse.h>        

std::mt19937 gen;
std::random_device rd;

float* generateMatrix(int M, int N, float sparsity) {
    std::uniform_real_distribution<float> entryDist(-1, 1);
    std::uniform_real_distribution<float> choiceDist(0, 1);

    float* out = new float[M * N];
    for (int i = 0; i < M * N; ++i) {
        if (choiceDist(gen) < sparsity)
            continue;
        out[i] = entryDist(gen);
    }

    return out;
}

void convertToCSR(float*& values, int*& rowOffsets, int*& colIndices, int& nnz, int M, int N, const float* input) {
    rowOffsets = new int[M + 1];

    for (int i = 0; i < M * N; ++i) {
        if (input[i] != 0)
            nnz += 1;
    }
    
    values = new float[nnz];
    colIndices = new int[nnz];

    int ind = 0;

    for (int i = 0; i < M; ++i) {
        for (int j = 0; j < N; ++j) {
            if (input[i * M + j] == 0)
                continue;
            values[ind] = input[i * M + j];
            colIndices[ind] = j;
        }
        rowOffsets[i + 1] = ind;
    }
}

void readDLMCMatrix(const std::string filename, float*& values, int*& rowOffsets, int*& colIndices, int& nnz, int& M, int& N) {
    std::ifstream matrixFile(filename);
    char comma;
    std::uniform_real_distribution<float> entryDist(-1, 1);

    matrixFile >> M >> comma >> N >> comma >> nnz;
    rowOffsets = new int[M + 1];
    colIndices = new int[nnz];
    values = new float[nnz];

    for (int i = 0; i < M + 1; ++i)
        matrixFile >> rowOffsets[i];
    for (int i = 0; i < nnz; ++i) {
        matrixFile >> colIndices[i];
        values[i] = entryDist(gen);
    }
}

int main() {
    gen = std::mt19937{rd()};
    hipStream_t stream;
    hipStreamCreate(&stream);

    int M = 4096;
    int N = 4096;
    int K = 4096;
    int sparsity = 0.8;
    
    float* A = generateMatrix(M, K, sparsity);
    float* B = generateMatrix(K, N, 0);
    
    float* valuesA;
    int* rowOffsetsA;
    int* colIndicesA;
    int nnz = 0;

    int* rowIndices = new int[M];

    for (int i = 0; i < M; ++i)
        rowIndices[i] = i;

    convertToCSR(valuesA, rowOffsetsA, colIndicesA, nnz, M, K, A);

    float* d_valuesA;
    hipMalloc(&d_valuesA, nnz * sizeof(float));
    int* d_colIndicesA;
    hipMalloc(&d_colIndicesA, N * sizeof(int));
    int* d_rowOffsetsA;
    hipMalloc(&d_rowOffsetsA, (M + 1) * sizeof(int));
    float* d_B;
    hipMalloc(&d_B, M * N * sizeof(float));
    
    int* d_rowIndices;
    hipMalloc(&d_rowIndices, M * sizeof(int));

    hipMemcpy(d_valuesA, valuesA, nnz * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_colIndicesA, colIndicesA, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_rowOffsetsA, rowOffsetsA, (M + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, M * N * sizeof(float), hipMemcpyHostToDevice);

    hipMemcpy(d_rowIndices, rowIndices, M * sizeof(int), hipMemcpyHostToDevice);

    float* d_C;
    hipMalloc(&d_C, M * N * sizeof(float));

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    sputnik::CudaSpmm(M, K, N, nnz, d_rowIndices, d_valuesA, d_rowOffsetsA, d_colIndicesA, d_B, d_C, stream);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    std::cout << milliseconds << std::endl;

    // code from cuSPARSE example
    
    hipsparseHandle_t     handle = NULL;
    hipsparseSpMatDescr_t matA;
    hipsparseDnMatDescr_t matB, matC;
    void*                dBuffer    = NULL;
    size_t               bufferSize = 0;
    float alpha           = 1.0f;
    float beta            = 0.0f;

    hipsparseCreate(&handle);
    hipsparseCreateCsr(&matA, M, K, nnz, d_rowOffsetsA, d_colIndicesA, d_valuesA,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F);

    hipsparseCreateDnMat(&matB, K, N, K, d_B, HIP_R_32F, HIPSPARSE_ORDER_COL);
    hipsparseCreateDnMat(&matC, M, N, M, d_C, HIP_R_32F, HIPSPARSE_ORDER_COL);
    hipsparseSpMM_bufferSize(     handle,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, matA, matB, &beta, matC, HIP_R_32F,
                                 HIPSPARSE_SPMM_ALG_DEFAULT, &bufferSize);
    hipMalloc(&dBuffer, bufferSize);
    hipsparseSpMM_preprocess(
                                 handle,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, matA, matB, &beta, matC, HIP_R_32F,
                                 HIPSPARSE_SPMM_ALG_DEFAULT, dBuffer);

    hipsparseSpMM(handle,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, matA, matB, &beta, matC, HIP_R_32F,
                                 HIPSPARSE_SPMM_ALG_DEFAULT, dBuffer);


    hipsparseDestroySpMat(matA);
    hipsparseDestroyDnMat(matB);
    hipsparseDestroyDnMat(matC);
    hipsparseDestroy(handle);

    hipFree(d_valuesA);
    hipFree(d_colIndicesA);
    hipFree(d_rowOffsetsA);
    hipFree(d_B);
    hipFree(d_rowIndices);
    hipFree(d_C);
    hipFree(dBuffer);

    delete A;
    delete B;
    delete valuesA;
    delete rowOffsetsA;
    delete colIndicesA;

    return 0;
}
