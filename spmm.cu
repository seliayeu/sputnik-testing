#include <cmath>
#include <iostream>
#include <fstream>
#include <sputnik/spmm/cuda_spmm.h>
#include <random>
#include <hip/hip_runtime_api.h> 
#include <hipsparse.h>        
#include "utils.h"

std::mt19937 gen;
std::random_device rd;

int main(int argc, char* argv[]) {
    gen = std::mt19937{rd()};

    std::string matrixPath;
    int sparsity;

    int M, N, K;

    float* valuesA;
    int* rowOffsetsA;
    int* colIndicesA;
    int nnz;

    if (argc == 2) {
        matrixPath = argv[1];
        readDLMCMatrix(matrixPath, valuesA, rowOffsetsA, colIndicesA, nnz, M, K);
    } else if (argc == 5) {
        M = std::stoi(argv[1]);
        K = std::stoi(argv[2]);
        N = std::stoi(argv[3]);;
        sparsity = std::stof(argv[4]);
        float* A = generateMatrix(M, K, sparsity);
        convertToCSR(valuesA, rowOffsetsA, colIndicesA, nnz, M, K, A);
        delete A;
    } else {
        std::cerr << "Invalid number of arguments." << std::endl;
    }
    
    N = M;
    float* B = generateMatrix(K, N, 0);

    float* d_valuesA;
    int* d_colIndicesA;
    int* d_rowOffsetsA;
    gpuErrchk(hipMalloc(&d_valuesA, nnz * sizeof(float)));
    gpuErrchk(hipMalloc(&d_colIndicesA, nnz * sizeof(int)));
    gpuErrchk(hipMalloc(&d_rowOffsetsA, (M + 1) * sizeof(int)));

    float* d_B;
    gpuErrchk(hipMalloc(&d_B, K * N * sizeof(float)));
    
    gpuErrchk(hipMemcpy(d_valuesA, valuesA, nnz * sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_colIndicesA, colIndicesA, nnz * sizeof(int), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_rowOffsetsA, rowOffsetsA, (M + 1) * sizeof(int), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_B, B, K * N * sizeof(float), hipMemcpyHostToDevice));

    float* d_C;
    gpuErrchk(hipMalloc(&d_C, M * N * sizeof(float)));

    hipEvent_t start, stop;

    // code from cuSPARSE example
    
    hipsparseHandle_t     handle = NULL;
    hipsparseSpMatDescr_t matA;
    hipsparseDnMatDescr_t matB, matC;
    void*                dBuffer    = NULL;
    size_t               bufferSize = 0;
    float alpha           = 1.0f;
    float beta            = 0.0f;

    hipsparseCreate(&handle);
    hipsparseCreateCsr(&matA, M, K, nnz, d_rowOffsetsA, d_colIndicesA, d_valuesA,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F);

    hipsparseCreateDnMat(&matB, K, N, K, d_B, HIP_R_32F, HIPSPARSE_ORDER_COL);
    hipsparseCreateDnMat(&matC, M, N, M, d_C, HIP_R_32F, HIPSPARSE_ORDER_COL);
    hipsparseSpMM_bufferSize(     handle,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, matA, matB, &beta, matC, HIP_R_32F,
                                 HIPSPARSE_SPMM_ALG_DEFAULT, &bufferSize);
    hipMalloc(&dBuffer, bufferSize);
    hipsparseSpMM_preprocess(
                                 handle,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, matA, matB, &beta, matC, HIP_R_32F,
                                 HIPSPARSE_SPMM_ALG_DEFAULT, dBuffer);

    gpuErrchk(hipEventCreate(&start));
    gpuErrchk(hipEventCreate(&stop));

    gpuErrchk(hipEventRecord(start));
    hipsparseStatus_t status;

    status = hipsparseSpMM(handle,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, matA, matB, &beta, matC, HIP_R_32F,
                                 HIPSPARSE_SPMM_ALG_DEFAULT, dBuffer);
    if (status != HIPSPARSE_STATUS_SUCCESS) {
        printf ("CUSPARSE kernel failed\n");
        return EXIT_FAILURE;
    }
    gpuErrchk(hipEventRecord(stop));
    gpuErrchk(hipEventSynchronize(stop));
    float milliseconds = 0;
    gpuErrchk(hipEventElapsedTime(&milliseconds, start, stop));

    std::cout << milliseconds << std::endl;

    hipsparseDestroySpMat(matA);
    hipsparseDestroyDnMat(matB);
    hipsparseDestroyDnMat(matC);
    hipsparseDestroy(handle);

    hipFree(d_valuesA);
    hipFree(d_colIndicesA);
    hipFree(d_rowOffsetsA);
    hipFree(d_B);
    hipFree(d_C);
    hipFree(dBuffer);

    delete B;
    delete valuesA;
    delete rowOffsetsA;
    delete colIndicesA;

    return 0;
}
